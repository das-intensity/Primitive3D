#include "hip/hip_runtime.h"
#include <optix.h>
#include "launch_parameters.h"

namespace prim3d {

extern "C" {
__constant__ RayCast::Params params;
}

// ray generation program
extern "C" __global__ void __raygen__rg() {
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    float3 ray_origin = params.ray_origins[idx.x];
    float3 ray_direction = params.ray_directions[idx.x];

    unsigned int p0, p1; // holder for the payload
    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        0.0f,                      // Min intersection distance
        1e16f,                     // Max intersection distance
        0.0f,                      // rayTime -- used for motion blur
        OptixVisibilityMask(255),  // Specify always visible
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0,  // SBT offset
        1,  // SBT stride
        0,  // missSBTIndex
        // payload
        p0,
        p1);

    // Hit position
    const int32_t ray_id = idx.x;
    params.hits[ray_id].w = int_as_float(p1);

    // If a triangle was hit, p0 is its index, otherwise p0 is -1.
    // Write out the triangle's normal if it (abuse the direction buffer).
    if ((int)p0 == -1) {
        return;
    }
    const float3 n = params.triangles[p0].normal();
    params.hits[ray_id].x = n.x;
    params.hits[ray_id].y = n.y;
    params.hits[ray_id].z = n.z;
}

// miss program
extern "C" __global__ void __miss__ms() {
    optixSetPayload_0((uint32_t)-1);
    optixSetPayload_1(__float_as_int(optixGetRayTmax()));
}

// closest-hit program
extern "C" __global__ void __closesthit__ch() {
    optixSetPayload_0(optixGetPrimitiveIndex());
    optixSetPayload_1(__float_as_int(optixGetRayTmax()));
}

}  // namespace prim3d
